#include <iostream>
#include <hip/hip_runtime.h>

// Kernel function to add two vectors
__global__ void vectorAdd(const float* A, const float* B, float* C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    // Size of vectors
    int numElements = 1024 * 1024;

    size_t size = numElements * sizeof(float);

    // Host input vectors
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize input vectors
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    // Device input vectors
    float* d_A = nullptr;
    float* d_B = nullptr;
    float* d_C = nullptr;

    // Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch the vectorAdd kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    for (int i = 0; i < numElements; ++i) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            std::cerr << "Error at element " << i << ": " << h_A[i] << " + " << h_B[i] << " != " << h_C[i] << std::endl;
            break;
        }
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    std::cout << "Vector addition complete!" << std::endl;

    return 0;
}